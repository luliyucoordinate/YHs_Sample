
#include <hip/hip_runtime.h>
#include <cstdint>
#include <cstdio>

const int DATA_SIZE = 32;

__global__ void smem_kernel_128(int4* ret) {
  __shared__ int4 mem[DATA_SIZE];

  // LDG.128
  for (int i = 0; i < DATA_SIZE; i++) {
    mem[i] = ret[i];
  }

  // STS.128
  for (int i = 0; i < DATA_SIZE; i++) {
    ret[i] = mem[i];
  }
}

__global__ void smem_kernel(int* ret) {
  __shared__ int mem[DATA_SIZE];

  // LDG.32
  for (int i = 0; i < DATA_SIZE; i++) {
    mem[i] = ret[i];
  }

  // STS.32
  for (int i = 0; i < DATA_SIZE; i++) {
    ret[i] = mem[i];
  }
}

// nvcc -cubin LDG_STS.cu
// nvdisasm -c LDG_STS.cubin > LDG_STS.sass

int main() {
  int4* d_ret_128;
  int* d_ret;
  hipMalloc(&d_ret_128, DATA_SIZE * sizeof(int4));
  hipMalloc(&d_ret, DATA_SIZE * sizeof(int));
  smem_kernel_128<<<1, 1>>>(d_ret_128);
  smem_kernel<<<1, 1>>>(d_ret);
  hipFree(d_ret);
  hipFree(d_ret_128);
}